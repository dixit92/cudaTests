#include "hip/hip_runtime.h"
/*
Performance Comparisions (HSA vs pure CPU) for standard CUDA programs

Floating point 2D Matrix Scaling
*/

#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdlib.h>
#include <Windows.h>
#include <stdio.h>

float executiontime = 0;
double cputime = 0;

double get_cpu_time();
void init_array(float **, const int, const int);
void print_array(float **, const int, const int);
void errcheck(hipError_t);
void printdevices();

//CUDA 2D Matrix Scaling kernel
__global__ void scaleKernel(float *d_in, float *d_out, int n, int m, float s)
{
	//Calculate row number of d_in and d_out element thread
	int r = blockIdx.y*blockDim.y + threadIdx.y;

	//Calculate col number of d_in and d_out element thread
	int c = blockIdx.x*blockDim.x + threadIdx.x;

	//perform scaling by scale factor s
	if ((r < m) && (c < n))
	{
		d_out[r*n + c] = s * d_in[r*n + c];
	}
}

//CUDA Kernel Init and Call
void ctScale(float **h_in, float **h_out, int n, int m, float s)
{

}

//CPU Calculation
void cpuScale(float **h_in, float **h_out, int n, int m, float s)
{
	int i, j;
	double start, stop;

	start = get_cpu_time();
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			h_out[i][j] = s * h_in[i][j];
		}
	}
	stop = get_cpu_time();

	cputime = stop - start;
}

int main()
{
	//Print Device
	printdevices();

	//Row/Column size - change depending on memory constraints
	const int rowsize = 5000;			
	const int colsize = 5000;

	//Scale factor
	const float scale = 2.25;

	//Initialize Arrays: Allocate memory
	int i;
	float *h_A[rowsize], *h_B[rowsize], *h_C[rowsize];
	for (i = 0; i < rowsize; i++)	h_A[i] = (float *)malloc(colsize * sizeof(float));
	for (i = 0; i < rowsize; i++)	h_B[i] = (float *)calloc(colsize,  sizeof(float));
	for (i = 0; i < rowsize; i++)	h_C[i] = (float *)calloc(colsize, sizeof(float));

	//Initialize h_A - randomized float elements
	printf("\nGenerating Random float point matrix...");
	init_array(h_A, rowsize, colsize);
	printf("\nGeneration complete.\n");

	/*Optional printing of elements, don't use for large row/col size*/
	/*
	printf("\nA:\n");
	print_array(h_A, rowsize, colsize);
	*/

	//CPU Calculation
	printf("\n\nStarting CPU Calculation...");
	cpuScale(h_A, h_C, rowsize, colsize, scale);
	printf("\nCPU Calculation complete.\n");
	
	/*Optional printing of elements, don't use for large row/col size*/
	/*
	printf("\nA * %f:\n", scale);
	print_array(h_C, rowsize, colsize);
	*/

	//Display performance comparision:
	printf("\nCUDA Execution time: %f", executiontime);
	printf("\nCPU Execution time %f", cputime);

	printf("\n\n");
	return 0;
}

//Windows CPU Time
double get_cpu_time(){
	FILETIME a, b, c, d;
	if (GetProcessTimes(GetCurrentProcess(), &a, &b, &c, &d) != 0){
		//  Returns total user time.
		//  Can be tweaked to include kernel times as well.
		return
			(double)(d.dwLowDateTime |
			((unsigned long long)d.dwHighDateTime << 32)) * 0.0000001;
	}
	else{
		return 0;
	}
}

//Populates arr of rows, cols with random elements
void init_array(float **arr, const int rows, const int cols)
{

	int i, j;
	for (i = 0; i < rows; i++)
	{
		for (j = 0; j < cols; j++)
		{
			arr[i][j] = (rand() / float(RAND_MAX));
		}
	}
}


//Prints arr of rows, cols
void print_array(float **arr, const int rows, const int cols)
{
	int i, j;
	for (i = 0; i < rows; i++)
	{
		for (j = 0; j < cols; j++)
		{
			printf("A[%d][%d]: %f ", i, j, arr[i][j]);

		}
		printf("\n");
	}
}

//Cuda error checking
void errcheck(hipError_t cerr)
{
	if (cerr != hipSuccess)
	{
		printf("%s in %s at %d\n", hipGetErrorString(cerr), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
}

//Prints information about cuda devices
void printdevices()
{
	int nDevices;
	hipError_t err;

	err = hipGetDeviceCount(&nDevices);
	errcheck(err);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}
}
